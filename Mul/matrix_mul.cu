#include <iostream>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <time.h>

#define N 8192 // Size of the matrices

__global__ void matrixMulKernel(half* d_A, half* d_B, float* d_C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    half sum = 0;

    if (row < width && col < width) {
        for (int k = 0; k < width; k++) {
            sum += d_A[row * width + k] * d_B[k * width + col];
        }
        d_C[row * width + col] = __half2float(sum);
    }
}

__global__ void matrixMulTensorCore(half* d_A, half* d_B, float* d_C, int width) {
    // Leading dimensions. Packed with no transpositions.
    int lda = width;
    int ldb = width;
    int ldc = width;

    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / 32;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y) / 8;

    // Declare the fragments
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::row_major> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> c_frag;

    nvcuda::wmma::fill_fragment(c_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < width; i += 16) {
        int aRow = warpM * 16;
        int aCol = i;
        int bRow = i;
        int bCol = warpN * 16;

        if (aRow < width && aCol < width && bRow < width && bCol < width) {
            // Load the inputs
            nvcuda::wmma::load_matrix_sync(a_frag, d_A + aRow * lda + aCol, lda);
            nvcuda::wmma::load_matrix_sync(b_frag, d_B + bRow * ldb + bCol, ldb);

            // Perform the matrix multiplication
            nvcuda::wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }
    }

    // Load in the current value of d_C, scale it by beta, and add this our result scaled by alpha
    int cRow = warpM * 16;
    int cCol = warpN * 16;
    // Store the output
    nvcuda::wmma::store_matrix_sync(d_C + cRow * ldc + cCol, c_frag, ldc, nvcuda::wmma::mem_row_major);
}

void matrixMul(half* h_A, half* h_B, float* h_C, int width) {
    int sizeIn = width * width * sizeof(half);
    int sizeOut = width * width * sizeof(float);
    half *d_A, *d_B;
    float *d_C;

    hipMalloc((void**)&d_A, sizeIn);
    hipMalloc((void**)&d_B, sizeIn);
    hipMalloc((void**)&d_C, sizeOut);

    hipMemcpy(d_A, h_A, sizeIn, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeIn, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                       (width + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMulTensorCore<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, width);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, sizeOut, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    half* h_A = (half*)malloc(N * N * sizeof(half));
    half* h_B = (half*)malloc(N * N * sizeof(half));
    float* h_C = (float*)malloc(N * N * sizeof(float));

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = __float2half(static_cast<float>(rand() % 10));
        h_B[i] = __float2half(static_cast<float>(rand() % 10));;
    }
    clock_t t = clock();
    matrixMul(h_A, h_B, h_C, N);
    t = clock() - t;

    /*
    printf("Result matrix:\n");
    for (int i = 0; i < 1; ++i) {
        for (int j = 0; j < 10; ++j) {
            printf("%.2f ", __half2float(h_C[i * N + j]));
        }
        printf("\n");
    }
    */

    double time_taken = ((double)t)/CLOCKS_PER_SEC;
    printf("The program took %f seconds to execute\n", time_taken);

    return 0;
}