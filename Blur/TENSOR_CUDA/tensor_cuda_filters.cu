#include "hip/hip_runtime.h"
#include <iostream>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../include/stb_image.h"
#include "../include/stb_image_write.h"
#include <stdio.h>
#include <time.h>

// librerias para CUDA
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>

// max 23 warps por bloque (exceed shared memory)
#define NUM_WARPS 8 // numero de warps por bloque maximo 32 (1024 threads)

#define WMMA_M 32
#define WMMA_N 8
#define WMMA_K 16

half * createFilter(int width)
{
        const float sigma = 2.f; // Standard deviation of the Gaussian distribution.

        const int middle = width / 2;
        float sum = 0.f;


        // Create convolution matrix
        float * aux = (float *)malloc(width*width*sizeof(float));
        half * res=(half *)malloc(width*width*sizeof(half));


        // Calculate filter sum first
        for (int r = -middle; r <= middle; ++r)
        {
                for (int c = -middle; c <= middle; ++c)
                {
                        // e (natural logarithm base) to the power x, where x is what's in the brackets
                        float weight = expf(-static_cast<float>(c * c + r * r) / (2.f * sigma * sigma));
                        int idx = (r + middle) * width + c + middle;

                        aux[idx] = weight;
                        sum += weight;
                }
        }

        // Normalize weight: sum of weights must equal 1
        float normal = 0.f;
        for (int i = 0; i < width * width; i++) {
                normal += (aux[i]);
        }
        for (int i = 0; i < width * width; i++) {
                res[i] = __float2half(aux[i] / (normal));
        }

        return res;
}

// Final matrix needs to be bigger than input matrix
void paddedFilter(half * filter, half * paddedFilter, int startSize, int finalSize) {
        int halfPadding = (finalSize - startSize) / 2;

        // Initialize the entire padded matrix with zeros
        for (int i = 0; i < finalSize * finalSize; i++) {
                paddedFilter[i] = __float2half(0.0f);
        }

        // Copy the original filter into the center of the padded matrix
        for (int row = 0; row < startSize; row++) {
                for (int col = 0; col < startSize; col++) {
                        int paddedRow = row + halfPadding;
                        int paddedCol = col + halfPadding;
                        paddedFilter[paddedRow * finalSize + paddedCol] = filter[row * startSize + col];
                }
        }
}

/*
Kernel de CUDA para realizar el desenfoque gaussiano
Estructura unidimensional de bloques (x para posicion)
Estructura unidimensional de threads (x para posicion y canal)
*/ 
__global__ void GaussianBlur(uint8_t* const blurredImage, const uint8_t* const rawImage, int width, int height, int channels, const half* filter, int filterWidth, int numFilters, float balance)
{        
        // Identificadores de threads
        int pos = blockIdx.x * blockDim.x + threadIdx.x;

        // Identificar thread dentro del warp
        int indexWarp = (threadIdx.x % (warpSize));

        // Identificar warp dentro del bloque
        int warpId = (threadIdx.x / warpSize);

        int temp = (pos / warpSize) * WMMA_M + indexWarp;
        // pixel y canal a tratar
        int x = (temp / channels) % width;
        int y = (temp / channels) / width;
        int canal = temp % channels;

        // Comprobar thread util
        if (x >= width || y >= height || canal >= channels){return;}

        // mitad ancho del filtro
        int halfFilterWidth = filterWidth / 2;

        // numero de valores de filtrado
        int filterSize = filterWidth * filterWidth;

        // reparto de warps
        float balancer = balance;
        int blockTensor = balancer * gridDim.x;
        
        // Implementacion TENSOR
        if (blockIdx.x < blockTensor) {
                
                // Definir estructura matrices
                nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::row_major> data;
                nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::col_major> mask;
                nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> result;
                
                // inicializar resultados a cero
                nvcuda::wmma::fill_fragment(result, 0.0f);

                // tamanho de cada matriz individual
                int offsetLocalMatrix = WMMA_M * WMMA_K * warpId;
                int offsetResultMatrix = WMMA_M * WMMA_N * warpId * (sizeof(float)/sizeof(half));

                // declarar matrices en memoria compartida
                // deben estar alineados
                extern __shared__ half sharedMemory[];
                
                // matriz de datos
                half* localMatrix = (half*)&sharedMemory[offsetLocalMatrix];
                
                // matriz de coeficientes
                half* filterMatrix = (half*)&sharedMemory[WMMA_M * WMMA_K * NUM_WARPS];
                
                // matriz de resultados
                float* resultMatrix = (float*)&sharedMemory[WMMA_M * WMMA_K * NUM_WARPS + WMMA_K * WMMA_N + offsetResultMatrix];

                // matriz intermedia
                half* interMatrix = (half*)&sharedMemory[WMMA_M * WMMA_K * NUM_WARPS + WMMA_K * WMMA_N + 
                                                         WMMA_M * WMMA_N * NUM_WARPS * (sizeof(float)/sizeof(half))];

                int pendingValues = filterSize;
                // Iterar por bloques en tamanho de warp
                for (int i = 0; i < filterSize; i+= WMMA_M) {
                        // posicion de inicio
                        int startIdx = i % filterWidth;

                        // comprobar si el filtro se ha cargado completamente
                        int toEnd = WMMA_K;
                        int filterY = (i / filterWidth);
                        while (toEnd > 0) { 
                                //calcular numero de valores hasta fin de fila
                                int restValues = filterWidth - startIdx;
                                //cargar datos en la matriz intermedia (primera fila)
                                for (int j = 0; j < restValues + warpSize - 1 - indexWarp; j+= warpSize) {
                                        //posicion a cargar
                                        int filterX = startIdx + j;
                                        //posicion absoluta en imagen
                                        int imageX = x - halfFilterWidth + filterX;
                                        int imageY = y - halfFilterWidth + filterY;
                                        //comprobacion de limites
                                        if ((imageY >= height) || (imageY == height - 1 && imageX >= width)) {break;}
                                        // Cargar el valor del pixel en interMatrix
                                        // TODO: indexar correctamente valores siguientes filas
                                        interMatrix[j + indexWarp] = (half)rawImage[((imageY * width + imageX) * channels) + canal];
                                }
                                filterY++;
                                startIdx = 0;
                                toEnd -= restValues;
                        }


                        __syncthreads();

                        //data matrix
                        // el indice no excede el numero de pixeles a cargar
                        if (indexWarp < WMMA_M) {
                                int temp = 0;

                                //iterar por todas las posiciones que se pueden cargar en la matriz
                                for (temp = 0; temp < min(WMMA_K, pendingValues); temp++) {
                                        //comprobacion de limites
                                        int filterX = ((temp + i) % filterWidth);
                                        int filterY = ((temp + i) / filterWidth);
                                        //obtener posicion pixel vecino
                                        int imageX = min(max(x + filterX - halfFilterWidth, 0), width - 1);
                                        int imageY = min(max(y + filterY - halfFilterWidth, 0), height - 1);

                                        //agregar vecino a matriz
                                        localMatrix[indexWarp * WMMA_K + temp] = 
                                                (half) rawImage[((imageY * width + imageX) * channels) + canal];
                                }
                                //rellenar con 0 en caso de necesitarlo
                                for (int j = temp; j < WMMA_K; j++) {
                                        localMatrix[indexWarp * WMMA_K + j] = 0;
                                        if (warpId == 0 && indexWarp < WMMA_N) {
                                                filterMatrix[indexWarp * WMMA_K + j] = 0;
                                        }
                                }
                        }
                        //agregar coeficiente en el filtro
                        if (indexWarp < min(WMMA_K, pendingValues) && warpId == 0) {
                                for (int j = 0; j < numFilters; j++) {
                                        filterMatrix[j * WMMA_K + indexWarp] = filter[j * filterSize + indexWarp + i];
                                }
                        }
                        __syncthreads();

                        // cargar en matriz data
                        nvcuda::wmma::load_matrix_sync(data, localMatrix, WMMA_K);
                        // cargar en matriz mask
                        nvcuda::wmma::load_matrix_sync(mask, filterMatrix, WMMA_K);
                        // ejecutar codigo en tensor
                        nvcuda::wmma::mma_sync(result, data, mask, result);
                        //reducir numero de valores faltantes
                        pendingValues -= WMMA_K;
                }
                // cargar resultado a matriz
                nvcuda::wmma::store_matrix_sync(resultMatrix, result, WMMA_N, nvcuda::wmma::mem_row_major);

                // almacenar resultados de vuelta en la memoria global
                if (indexWarp < WMMA_M) {
                        //ejemplo de almacenamiento de resultados
                        for (int i = 0; i < numFilters; i++) {
                                blurredImage[((y * width + x) * channels) + canal + (i * width * height * channels)] = (uint8_t) resultMatrix[indexWarp * WMMA_N + i];
                        }
                }
        } else {     
                for (int i = 0; i < numFilters; i++) {   
                        //Implementacion CUDA
                        // pixel desenfocado
                        half blurredPixel = 0;
                        // Calcular el pixel desenfocado
                        for (int filterY = -halfFilterWidth; filterY <= halfFilterWidth; filterY++) {
                                for (int filterX = -halfFilterWidth; filterX <= halfFilterWidth; filterX++) {
                                        
                                        //comprobacion de limites
                                        int imageX = min(max(x + filterX, 0), width - 1);
                                        int imageY = min(max(y + filterY, 0), height - 1);

                                        // Calcular el indice del filtro
                                        int filterIndex = (filterY + halfFilterWidth) * filterWidth + (filterX + halfFilterWidth);
                                        
                                        // Pixel de la imagen a tratar
                                        half pixel = (half) rawImage[((imageY * width + imageX) * channels) + canal];
                                        blurredPixel += pixel * filter[filterIndex + i * filterSize];
                                }
                        }
                        blurredImage[((y * width + x) * channels) + canal + (i * width * height * channels)] = (uint8_t) __half2float(blurredPixel);
                }
        }        
}

// Main entry into the application
int main(int argc, char** argv)
{

	char * imagePath;
	char * outputPath;
	
	int height, width, bpp, channels=4, filterWidth, numFilters;
	uint8_t * originalImage, * blurredImage;
        float balance;

	if (argc > 4)
	{
		imagePath = argv[1];
		outputPath = argv[2];
                filterWidth = atoi(argv[3]);
                numFilters = atoi(argv[4]);
                balance = atof(argv[5]);
	}
	else
	{
		printf("Please provide input and output image files, filter size and number of it as arguments to this application.\n");
		exit(1);
	}
        
        //PRINT TITLE
        printf(
                " ░▒▓██████▓▒░ ░▒▓██████▓▒░░▒▓█▓▒░░▒▓█▓▒░░▒▓███████▓▒░▒▓███████▓▒░▒▓█▓▒░░▒▓██████▓▒░░▒▓███████▓▒░ \n"  
                "░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░     ░▒▓█▓▒░      ░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░\n"
                "░▒▓█▓▒░      ░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░     ░▒▓█▓▒░      ░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░\n" 
                "░▒▓█▓▒▒▓███▓▒░▒▓████████▓▒░▒▓█▓▒░░▒▓█▓▒░░▒▓██████▓▒░░▒▓██████▓▒░░▒▓█▓▒░▒▓████████▓▒░▒▓█▓▒░░▒▓█▓▒░\n" 
                "░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░      ░▒▓█▓▒░     ░▒▓█▓▒░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░\n" 
                "░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░      ░▒▓█▓▒░     ░▒▓█▓▒░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░\n" 
                " ░▒▓██████▓▒░░▒▓█▓▒░░▒▓█▓▒░░▒▓██████▓▒░░▒▓███████▓▒░▒▓███████▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░▒▓█▓▒░░▒▓█▓▒░\n"
                );
        
        //crear filtros
        
        // matriz que guarda los filtros
        half * filters = (half *)malloc(filterWidth * filterWidth * numFilters * sizeof(half));

        // iterar por los filtros a crear
        int filterAux = filterWidth;
        for (int i = 0; i < numFilters; i++) {
                // crear filtro
                half * filter = createFilter(filterAux);
                paddedFilter(filter, &filters[i * filterWidth * filterWidth], filterAux, filterWidth);
                if (filterAux > 1) {
                        filterAux -= 2;
                }
        }

	//Read the image
	originalImage = stbi_load(imagePath, &width, &height, &bpp, channels);
	
	if(originalImage==NULL) printf("Could not load image file: %s\n",imagePath);
        
	blurredImage=(uint8_t *)malloc(width*height*channels*numFilters*sizeof(uint8_t));
	printf("Width:%d, Height:%d Size(in Bytes):%lu\n", width, height, (long unsigned int) width*height*bpp*channels);

        // Definir punteros para la memoria de la GPU
        uint8_t *d_originalImage, *d_blurredImage;
        half *d_filter;

        // Reservar memoria en la GPU para la imagen original y la imagen final
        hipMalloc((void**)&d_originalImage, width * height * channels * sizeof(uint8_t));
        hipMalloc((void**)&d_blurredImage, width * height * channels * numFilters * sizeof(uint8_t));
        hipMalloc((void**)&d_filter, filterWidth * filterWidth * numFilters * sizeof(half));

        // Copiar la imagen original y filtro desde la memoria del host a la memoria de la GPU
        hipMemcpy(d_originalImage, originalImage, width * height * channels * sizeof(uint8_t), hipMemcpyHostToDevice);
        hipMemcpy(d_filter, filters, filterWidth * filterWidth * numFilters * sizeof(half), hipMemcpyHostToDevice);

        //procedimiento
        int threadsPerBlock = NUM_WARPS * 32;
        dim3 blockDim(threadsPerBlock);
        dim3 gridDim((width * height) / ((threadsPerBlock) / channels) + 1);

        // espacio de memoria compartida
        size_t sharedMemorySize = (WMMA_M * WMMA_K) * NUM_WARPS * sizeof(half) + 
                                  (WMMA_N * WMMA_K) * sizeof(half) +
                                  (WMMA_M * WMMA_N) * NUM_WARPS * 2 * sizeof(float);

        // Iniciar el temporizador
        clock_t t = clock();

        GaussianBlur<<<gridDim, blockDim, sharedMemorySize>>>(d_blurredImage, d_originalImage, width, height, channels, d_filter, filterWidth, numFilters, balance);

        hipDeviceSynchronize();
        
        //time
        t = clock() - t;

        // Copiar la imagen final desde la memoria de la GPU a la memoria del host
        hipMemcpy(blurredImage, d_blurredImage, width * height * channels * numFilters * sizeof(uint8_t), hipMemcpyDeviceToHost);

        // Guardar la imagen desenfocada en un archivo
        char * outputPathAux = (char *)malloc(strlen(outputPath) + 10 * sizeof(char));
        for (int i = 0; i < numFilters; i++) {
                strcpy(outputPathAux, outputPath);
                sprintf(outputPathAux + strlen(outputPath), "_%d.jpg", i);
	        stbi_write_jpg(outputPathAux, width, height, 4, &blurredImage[i * width * height * channels], 100);
        }

        // Liberar la memoria de la GPU
        hipFree(d_originalImage);
        hipFree(d_blurredImage);
        hipFree(d_filter);
        // Liberar la memoria del host
        free(originalImage);
        free(blurredImage);
        free(filters);

	printf("Done!\n");

        double time_taken = ((double)t)/CLOCKS_PER_SEC;
        printf("The program took %f seconds to execute\n", time_taken);
	return 0;
}
